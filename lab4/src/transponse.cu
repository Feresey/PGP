#include "hip/hip_runtime.h"
#include <cmath>

#include <thrust/detail/raw_pointer_cast.h>

#include "helpers.cuh"

__global__ void transponse_kernel(
    double* out, const double* A,
    const uint32_t n, const uint32_t m)
{
    const uint tid_x = threadIdx.x,
               tid_y = threadIdx.y;
    const uint idx = blockDim.x * blockIdx.x + threadIdx.x,
               idy = blockDim.y * blockIdx.y + threadIdx.y;
    const uint idx_T = blockDim.x * blockIdx.x + threadIdx.y,
               idy_T = blockDim.y * blockIdx.y + threadIdx.x;

    __shared__ double shared[32][32 + 1];

    if (idx < n && idy < m) {
        shared[tid_x][tid_y] = A[idx * m + idy];
        // printf("idx  : %d\tidy  : %d\t%lf\n", idx, idy, shared[tid_x][tid_y]);
    }

    __syncthreads();
    if (idy < n && idx < m) {
        // printf("idx_t: %d\tidy_t: %d\t%lf\n", idx_T, idy_T, shared[tid_y][tid_x]);
        out[idy_T * n + idx_T] = shared[tid_y][tid_x];
    }
}

template <class T>
T div_up(T a, T b) { return (a - 1) / b + 1; }

dev_matrix transponse(const dev_matrix& A, const uint32_t n, const uint32_t m)
{
    const double* raw = thrust::raw_pointer_cast(&A[0]);
    dev_matrix res(m * n);
    double* res_raw = thrust::raw_pointer_cast(&res[0]);
    const dim3 blocks = dim3(div_up<uint>(n, 32), div_up<uint>(m, 32));
    const dim3 threads = dim3(32, 32);

    START_KERNEL((transponse_kernel<<<blocks, threads>>>(res_raw, raw, n, m)));

    // show_matrix(stderr, A, n, m);
    // show_matrix(stderr, A_trans, m, n);

    return res;
}