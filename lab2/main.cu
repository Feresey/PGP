#include "hip/hip_runtime.h"
#include <stdio.h>

#include "serialize.h"

#define CSC(call)                                              \
    do {                                                       \
        if (call != hipSuccess) {                             \
            fprintf(stderr,                                    \
                "ERROR in %s:%d. Message: %s\n",               \
                __FILE__, __LINE__, hipGetErrorString(call)); \
            exit(0);                                           \
        }                                                      \
    } while (0)

texture<uchar4, 2, hipReadModeElementType> tex;

__global__ void kernel(uchar4* out, uint32_t w, uint32_t h)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    for (int x = idx; x < w; x += offsetx) {
        for (int y = idy; y < h; y += offsety) {
            out[x + y * w] = uchar4(tex2D(tex, x, y));
        }
    }
}


int main(int argc, char** argv)
{
    int blocks = 1;
    int threads = 32;
#ifdef BENCHMARK
    for (int i = 1; i < argc; i += 2) {
        if (strcmp(argv[i], "-blocks") == 0) {
            blocks = atoi(argv[i + 1]);
        } else if (strcmp(argv[i], "-threads") == 0) {
            threads = atoi(argv[i + 1]);
        }
    }
#endif

    char input[100], output[200];

    scanf("%s", input);
    scanf("%s", output);

    FILE* in = fopen(input, "rb");

    uint32_t* data;
    uint32_t w, h;
    uint32_t err;
    err = read_image(in, &data, &w, &h);
    if (err != 0) {
        printf("ERROR in %s:%d scan image: %d", __FILE__, __LINE__, err);
        exit(0);
    }
    fclose(in);

    hipArray* arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, w, h));
    CSC(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * h * w, hipMemcpyHostToDevice));

    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.channelDesc = ch;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = false;

    CSC(hipBindTextureToArray(tex, arr, ch));
    uchar4* dev_data;
    CSC(hipMalloc(&dev_data, sizeof(uchar4) * h * w));

#ifdef BENCHMARK
    hipEvent_t start, end;
    CSC(hipEventCreate(&start));
    CSC(hipEventCreate(&end));
    CSC(hipEventRecord(start));

    fprintf(stderr, "blocks = %d\nthreads = %d\n", blocks, threads);
#endif

    kernel<<<dim3(blocks, blocks), dim3(threads, threads)>>>(dev_data, w, h);
    CSC(hipGetLastError());

#ifdef BENCHMARK
    CSC(hipEventRecord(end));
    CSC(hipEventSynchronize(end));
    float t;
    CSC(hipEventElapsedTime(&t, start, end));
    CSC(hipEventDestroy(start));
    CSC(hipEventDestroy(end));

    fprintf(stderr, "time = %010.6f\n", t);
#endif

    CSC(hipMemcpy(data, dev_data, sizeof(uchar4) * h * w, hipMemcpyDeviceToHost));

    FILE* out = fopen(output, "wb");
    err = write_image(out, data, w, h);
    if (err != 0) {
        printf("ERROR in %s:%d write image: %d", __FILE__, __LINE__, err);
    }
    fclose(out);

    CSC(hipUnbindTexture(tex));
    CSC(hipFreeArray(arr));
    CSC(hipFree(dev_data));
    free(data);
}
