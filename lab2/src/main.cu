#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

#include "serialize.h"

#define CSC(kal)                                               \
    do {                                                       \
        auto call = kal;                                       \
        if (call != hipSuccess) {                             \
            fprintf(stderr,                                    \
                "ERROR in %s:%d. Message: %s\n",               \
                __FILE__, __LINE__, hipGetErrorString(call)); \
            exit(0);                                           \
        }                                                      \
    } while (0)

texture<uchar4, 2, hipReadModeElementType> tex;

__device__ int4 uchar4sum(uchar4 a, uchar4 b, uchar4 c)
{
    int4 res;
    res.x = int(a.x) + int(b.x) + int(c.x);
    res.y = int(a.y) + int(b.y) + int(c.y);
    res.z = int(a.z) + int(b.z) + int(c.z);
    res.w = 0;
    return res;
}

__device__ int4 int4sub(int4 a, int4 b)
{
    int4 res;
    res.x = a.x - b.x;
    res.y = a.y - b.y;
    res.z = a.z - b.z;
    res.w = 0;
    return res;
}

#define norm(u) (0.299 * float(u.x) + 0.587 * float(u.y) + 0.114 * float(u.z))
#define meanless(a, b) sqrtf(float(a * a) + float(b * b))

__device__ float prewitt(uchar4* z)
{
    int4 up = uchar4sum(z[0], z[1], z[2]);
    int4 down = uchar4sum(z[6], z[7], z[8]);

    int4 right = uchar4sum(z[0], z[3], z[6]);
    int4 left = uchar4sum(z[2], z[5], z[8]);

    int4 g_x = int4sub(down, up);
    int4 g_y = int4sub(left, right);

    return meanless(norm(g_x), norm(g_y));
}

__global__ void kernel(uchar4* out, uint32_t w, uint32_t h)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    uchar4 z[9];
    int left, right, top, bottom;
    for (int x = idx; x < w; x += offsetx) {
        for (int y = idy; y < h; y += offsety) {

            left = x - 1;
            right = x + 1;
            top = y - 1;
            bottom = y + 1;

            z[0] = tex2D(tex, left, top);
            z[1] = tex2D(tex, x, top);
            z[2] = tex2D(tex, right, top);
            z[3] = tex2D(tex, left, y);
            z[4] = tex2D(tex, x, y);
            z[5] = tex2D(tex, right, y);
            z[6] = tex2D(tex, left, bottom);
            z[7] = tex2D(tex, x, bottom);
            z[8] = tex2D(tex, right, bottom);

            float res = prewitt(z);
            unsigned char res_byte = res;
            if (res < 0) {
                printf("ERROR: ты обосрался: %f\n", res);
            }
            if (res > 255) {
                res_byte = 255;
            }
            out[x + y * w] = make_uchar4(res_byte, res_byte, res_byte, 0);
        }
    }
}

#ifdef BENCHMARK
int main(int argc, char** argv)
#else
int main()
#endif
{
    unsigned int blocks = 1;
    unsigned int threads = 8;
#ifdef BENCHMARK
    for (int i = 1; i < argc; i += 2) {
        if (strcmp(argv[i], "-blocks") == 0) {
            blocks = atoi(argv[i + 1]);
        } else if (strcmp(argv[i], "-threads") == 0) {
            threads = atoi(argv[i + 1]);
        }
    }
#endif

    char input[255], output[255];

    scanf("%s", input);
    scanf("%s", output);

    FILE* in = fopen(input, "rb");
    if (in == NULL || ferror(in)) {
        perror(NULL);
        printf("ERROR opening input file: %s\n", input);
        exit(0);
    }

    uint32_t* data;
    uint32_t w, h;
    int err;
    err = read_image(in, &data, &w, &h);
    if (err != 0) {
        printf("ERROR in %s:%d scan image: %d", __FILE__, __LINE__, err);
        exit(0);
    }
    fclose(in);

    hipArray* arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, w, h));
    CSC(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * h * w, hipMemcpyHostToDevice));

    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.channelDesc = ch;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = false;

    CSC(hipBindTextureToArray(tex, arr, ch));
    uchar4* dev_data;
    CSC(hipMalloc(&dev_data, sizeof(uchar4) * h * w));

#ifdef BENCHMARK
    hipEvent_t start, end;
    CSC(hipEventCreate(&start));
    CSC(hipEventCreate(&end));
    CSC(hipEventRecord(start));

    fprintf(stderr, "blocks = %d\nthreads = %d\n", blocks, threads);
#endif

    kernel<<<dim3(blocks, blocks), dim3(threads, threads)>>>(dev_data, w, h);
    CSC(hipGetLastError());

#ifdef BENCHMARK
    CSC(hipEventRecord(end));
    CSC(hipEventSynchronize(end));
    float t;
    CSC(hipEventElapsedTime(&t, start, end));
    CSC(hipEventDestroy(start));
    CSC(hipEventDestroy(end));

    fprintf(stderr, "time = %010.6f\n", t);
#endif

    CSC(hipMemcpy(data, dev_data, sizeof(uchar4) * h * w, hipMemcpyDeviceToHost));

    FILE* out = fopen(output, "wb");
    err = write_image(out, data, w, h);
    if (err != 0) {
        printf("ERROR in %s:%d write image: %d", __FILE__, __LINE__, err);
    }
    fclose(out);

    CSC(hipUnbindTexture(tex));
    CSC(hipFreeArray(arr));
    CSC(hipFree(dev_data));
    free(data);
}
