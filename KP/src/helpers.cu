#include "helpers.cuh"

void setDevice(int device_id)
{
    CUDA_ERR(hipSetDevice(device_id));
}

void getDeviceCount(int* device_count)
{
    CUDA_ERR(hipGetDeviceCount(device_count));
}
