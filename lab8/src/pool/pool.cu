#include "pool.hpp"

void GPU_pool::init_devices(int max_dim)
{
    std::vector<double> temp(max_dim * max_dim);

    for (auto device_it = devices.begin(); device_it != devices.end(); ++device_it) {
        Elem& device = *device_it;
        int device_id = int(std::distance(devices.begin(), device_it));

        const int buffer_size = device.grid.cells_per_block() * sizeof(double);

        CUDA_ERR(hipMalloc(&device.gpu_data, buffer_size));
        CUDA_ERR(hipMalloc(&device.gpu_data_next, buffer_size));
        CUDA_ERR(hipMalloc(&device.gpu_buffer, temp.size() * sizeof(double)));

        for (int k = -1; k <= device.grid.bsize.z; ++k) {
            for (int j = -1; j <= device.grid.bsize.y; ++j) {
                for (int i = -1; i <= device.grid.bsize.x; ++i) {
                    const mydim3<int> cell = { i, j, k };
                    const dim3_type type = layer_tag_to_dim3_type(split_type);

                    mydim3<int> abused_cell = cell;
                    abused_cell[type] = cell[type] + device.grid.bsize[type] * device_id;

                    temp[device.grid.cell_absolute_id(cell)] = buffer[device.grid.cell_absolute_id(abused_cell)];
                }
            }
        }

        CUDA_ERR(hipMemcpy(device.gpu_data, temp.data(), buffer_size, hipMemcpyHostToDevice));
    }
}

void GPU_pool::free_devices()
{
    for (Elem& device : devices) {
        CUDA_ERR(hipFree(device.gpu_data));
        CUDA_ERR(hipFree(device.gpu_data_next));
        CUDA_ERR(hipFree(device.gpu_buffer));
    }
}
