#include "hip/hip_runtime.h"
#include <assert.h>
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>

#include "helpers.cuh"

#define BLOCK_SIZE 1024
#define WARP_SIZE 32

#define idx(ind) \
    ((WARP_SIZE + 1) * (ind / WARP_SIZE) + (ind % WARP_SIZE))

int next_multiple(int n, int m)
{
    int r = n % m;
    if (r == 0)
        return n;
    return n + (m - r);
}

__device__ void conditional_swap(int* x, int* y)
{
    int x_val = *x;
    int y_val = *y;
    if (x_val > y_val) {
        *y = x_val;
        *x = y_val;
    }
}

__global__ void int_memset(int* dev_arr, int n, int val)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = gridDim.x * blockDim.x;

    for (int i = idx; i < n; i += offset)
        dev_arr[i] = val;
}

__global__ void sort_blocks(int* dev_arr)
{
    __shared__ int sarr[BLOCK_SIZE + (BLOCK_SIZE / WARP_SIZE) + 1];

    sarr[idx(threadIdx.x)] = dev_arr[threadIdx.x + blockIdx.x * BLOCK_SIZE];
    int second_half_idx = threadIdx.x + BLOCK_SIZE / 2;
    sarr[idx(second_half_idx)] = dev_arr[threadIdx.x + blockIdx.x * BLOCK_SIZE + BLOCK_SIZE / 2];

    if (threadIdx.x == 0)
        sarr[idx(BLOCK_SIZE)] = INT_MAX;

    __syncthreads();

    int swap1_idx1 = 2 * threadIdx.x;
    int swap1_idx2 = 2 * threadIdx.x + 1;
    int swap2_idx2 = 2 * threadIdx.x + 2;

    for (int i = 0; i < BLOCK_SIZE; ++i) {
        conditional_swap(sarr + idx(swap1_idx1), sarr + idx(swap1_idx2));
        __syncthreads();

        conditional_swap(sarr + idx(swap1_idx2), sarr + idx(swap2_idx2));
        __syncthreads();
    }

    __syncthreads();

    dev_arr[threadIdx.x + blockIdx.x * BLOCK_SIZE] = sarr[idx(threadIdx.x)];
    dev_arr[threadIdx.x + blockIdx.x * BLOCK_SIZE + BLOCK_SIZE / 2] = sarr[idx(second_half_idx)];
}

__global__ void merge(int* dev_arr, int iter, int type)
{
    __shared__ int sarr[BLOCK_SIZE + (BLOCK_SIZE / WARP_SIZE)];

    sarr[idx(threadIdx.x)] = dev_arr[threadIdx.x + blockIdx.x * BLOCK_SIZE];
    int load_second_half_idx = BLOCK_SIZE - threadIdx.x - 1;
    sarr[idx(load_second_half_idx)] = dev_arr[threadIdx.x + blockIdx.x * BLOCK_SIZE + BLOCK_SIZE / 2];

    __syncthreads();

    for (int stride = BLOCK_SIZE / 2; stride > 0; stride /= 2) {
        int i = threadIdx.x / stride;
        int j = threadIdx.x % stride;

        int swap_idx1 = 2 * stride * i + j;
        int swap_idx2 = 2 * stride * i + j + stride;

        __syncthreads();
        conditional_swap(sarr + idx(swap_idx1), sarr + idx(swap_idx2));
    }

    __syncthreads();

    dev_arr[threadIdx.x + blockIdx.x * BLOCK_SIZE] = sarr[idx(threadIdx.x)];
    int store_second_half_idx = threadIdx.x + BLOCK_SIZE / 2;
    ;
    dev_arr[threadIdx.x + blockIdx.x * BLOCK_SIZE + BLOCK_SIZE / 2] = sarr[idx(store_second_half_idx)];
}

void block_odd_even_sort(int* arr, int n)
{
    int dev_n = next_multiple(n, BLOCK_SIZE);
    int n_blocks = dev_n / BLOCK_SIZE;

    int* dev_arr;
    CSC(hipMalloc(&dev_arr, dev_n * sizeof(int)));
    CSC(hipMemcpy(dev_arr, arr, n * sizeof(int), hipMemcpyHostToDevice));

    START_KERNEL((int_memset<<<1, BLOCK_SIZE>>>(dev_arr + n, (dev_n - n), INT_MIN)));

    START_KERNEL((sort_blocks<<<n_blocks, BLOCK_SIZE / 2>>>(dev_arr)));

    if (n_blocks == 1) {
        CSC(hipMemcpy(arr, dev_arr + (dev_n - n), n * sizeof(int), hipMemcpyDeviceToHost));
        CSC(hipFree(dev_arr));
        return;
    }

    for (int iter = 0; iter < n_blocks; ++iter) {
        START_KERNEL((merge<<<n_blocks - 1, BLOCK_SIZE / 2>>>(dev_arr + BLOCK_SIZE / 2, iter, 0)));
        START_KERNEL((merge<<<n_blocks, BLOCK_SIZE / 2>>>(dev_arr, iter, 1)));
    }

    CSC(hipMemcpy(arr, dev_arr + (dev_n - n), n * sizeof(int), hipMemcpyDeviceToHost));
    CSC(hipFree(dev_arr));
}

int main()
{
    uint32_t size = scan_4();

    int* arr = (int*)malloc(size * sizeof(int));
    for (uint32_t i = 0; i < size; ++i) {
        arr[i] = int(scan_4());
    }

    block_odd_even_sort(arr, size);

    print_arr(stdout, arr, size);

    free(arr);
    return 0;
}