#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

#include "helpers.h"

#define EPS 1e-3
#define IN_EPS(EQ) (abs(EQ) < EPS)

// да, костыль
int blocks = 1;
int threads = 32;

__device__ __constant__ float4 dev_centers[500];

__device__ float distance(const uchar4& a, const float4& b)
{
    // printf("%f %f %f\n", b.x, b.y, b.z);
    float x = b.x - a.x,
          y = b.y - a.y,
          z = b.z - a.z;
    return x * x + y * y + z * z;
}

__device__ int calc_best_distance(const uchar4& point, const int n_classes)
{
    int best_class = 255;
    float best_distance = 690420;

    for (int i = 0; i < n_classes; i++) {
        float curr_distance = distance(point, dev_centers[i]);
        // printf("compare %f<>%f\n", curr_distance, best_distance);
        if (curr_distance < best_distance) {
            best_class = i;
            best_distance = curr_distance;
        }
    }

    return best_class;
}

__device__ float norm(float4 a, float4 b)
{
    float x = a.x - b.x,
          y = a.y - b.y,
          z = a.z - b.z;
    return x * x + y * y + z * z;
}

// классификация пикселей по текущим центрам групп.
__global__ void kernel(
    uchar4* data, int n,
    float4* new_centers, ulonglong4* cache, int n_classes,
    unsigned long long* equal)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;

    for (int i = id; i < n; i += offset) {
        data[i].w = calc_best_distance(data[i], n_classes);

        // вычисление новых центров классов
        // суммирование значений пикселей по классам
        uchar4 elem = data[i];
        ulonglong4* cache_elem = &cache[elem.w];
        atomicAdd(&cache_elem->x, elem.x);
        atomicAdd(&cache_elem->y, elem.y);
        atomicAdd(&cache_elem->z, elem.z);
        atomicAdd(&cache_elem->w, 1);
    }

    __syncthreads();
    // присваивание новых значений центров классов.
    for (int i = id; i < n_classes; i += offset) {
        ulonglong4 cache_elem = cache[i];
        float4 elem = make_float4(
            float(cache_elem.x) / cache_elem.w,
            float(cache_elem.y) / cache_elem.w,
            float(cache_elem.z) / cache_elem.w,
            0.0f);
        new_centers[i] = elem;

        // условие сходимости -- центры не изменились
        float4 old = dev_centers[i];
        if (norm(old, elem) > EPS) {
            // printf("%f %f %f <> %f %f %f\n", old.x, old.y, old.z, elem.x, elem.y, elem.z);
            atomicAdd(equal, 1);
        }
    }
}

typedef struct {
    int x, y;
} Center;

void launch_k_means(uchar4* host_data, const int w, const int h, const Center* start_centers, const int n_classes)
{
    const int n = h * w;

    float4* dev_next_centers;
    uchar4* dev_data;
    ulonglong4* dev_cache;

    CSC(hipMalloc(&dev_next_centers, sizeof(float4) * n));
    CSC(hipMalloc(&dev_cache, sizeof(ulonglong4) * n));

    CSC(hipMalloc(&dev_data, sizeof(uchar4) * n));
    CSC(hipMemcpy(dev_data, host_data, sizeof(uchar4) * n, hipMemcpyHostToDevice));

    {
        float4* tmp_centers;
        // значения указанных пикселей.
        tmp_centers = (float4*)malloc(sizeof(uchar4) * n);
        for (int i = 0; i < n_classes; i++) {
            uchar4 elem = host_data[start_centers[i].y * w + start_centers[i].x];
            tmp_centers[i] = make_float4(elem.x, elem.y, elem.z, 0.0f);
        }
        // printf("init\n");
        // for (int i = 0; i < n_classes; i++) {
        //     uchar4 m = tmp_centers[i];
        // printf("%d %d %d\n", m.x, m.y, m.z);
        // }
        // printf("\n\n");
        CSC(hipMemcpy(dev_next_centers, tmp_centers, sizeof(float4) * n, hipMemcpyHostToDevice));
        free(tmp_centers);
    }

    unsigned long long equal = 0,
                       *dev_equal;
    CSC(hipMalloc(&dev_equal, sizeof(unsigned long long)));

    int killme = 5;
    while (killme--) {
        CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_centers), dev_next_centers, sizeof(float4) * n_classes, 0, hipMemcpyDeviceToDevice));

        // CSC(hipMemcpy(host_data, dev_data, sizeof(uchar4) * n, hipMemcpyDeviceToHost));
        // printf("===\n");
        // for (int x = 0; x < w; x++) {
        //     for (int y = 0; y < h; y++) {
        //         printf("%d ", x * w + y);
        //         uchar4 p = host_data[x * w + y];
        //         printf("%02x%02x%02x %d ", p.x, p.y, p.z, p.w);
        //     }
        //     printf("\n");
        // }
        // printf("===\n");
        // for (int i = 0; i < n; i++) {
        //     printf("%d ", i);
        //     uchar4 p = host_data[i];
        //     printf("%d %d %d %d\n", p.x, p.y, p.z, p.w);
        // }
        // printf("===\n");

        CSC(hipMemset(dev_equal, 0, sizeof(unsigned long long)));
        CSC(hipMemset(dev_cache, 0, sizeof(ulonglong4) * n_classes));

        START_KERNEL((kernel<<<blocks, threads>>>(
            dev_data, n,
            dev_next_centers, dev_cache, n_classes,
            dev_equal)));

        CSC(hipMemcpy(&equal, dev_equal, sizeof(unsigned long long), hipMemcpyDeviceToHost));
        // printf("equal: %llu\n", equal);
        if (equal == 0) {
            break;
        }
    }

    CSC(hipFree(dev_equal));
    CSC(hipMemcpy(host_data, dev_data, sizeof(uchar4) * n, hipMemcpyDeviceToHost));
    CSC(hipFree(dev_data));
    CSC(hipFree(dev_next_centers));
    CSC(hipFree(dev_cache));
}

int main()
{
#ifdef BENCHMARK
    for (int i = 1; i < argc; i += 2) {
        if (strcmp(argv[i], "-blocks") == 0) {
            blocks = atoi(argv[i + 1]);
        } else if (strcmp(argv[i], "-threads") == 0) {
            threads = atoi(argv[i + 1]);
        }
    }
#endif

    char input[100], output[100];

    scanf("%s", input);
    scanf("%s", output);

    int n_classes;
    Center* centers;
    scanf("%d", &n_classes);
    centers = (Center*)malloc(n_classes * sizeof(Center));
    for (int i = 0; i < n_classes; i++) {
        scanf("%d", &centers[i].x);
        scanf("%d", &centers[i].y);
    }

    FILE* in = fopen(input, "rb");
    if (ferror(in)) {
        printf("ERROR opening input file: %s\n", input);
        exit(0);
    }

    uchar4* data;
    uint32_t w, h;
    uint32_t err;
    err = read_image(in, &data, &w, &h);
    if (err != 0) {
        printf("ERROR in %s:%d scan image: %d", __FILE__, __LINE__, err);
        exit(0);
    }
    fclose(in);

    launch_k_means(data, h, w, centers, n_classes);

    FILE* out = fopen(output, "wb");
    err = write_image(out, data, w, h);
    if (err != 0) {
        printf("ERROR in %s:%d write image: %d", __FILE__, __LINE__, err);
    }
    fclose(out);

    free(data);
    return 0;
}