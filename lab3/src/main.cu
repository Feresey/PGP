#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

#include "helpers.cuh"

#define EPS 1e-3
#define IN_EPS(EQ) (abs(EQ) < EPS)

// да, костыль
dim3 blocks = 1;
dim3 threads = 32;

__device__ __constant__ float4 dev_centers[500];

// расстояние между центром и пикселем
__device__ float distance(const uchar4& a, const float4& b)
{
    // printf("%f %f %f\n", b.x, b.y, b.z);
    float x = b.x - float(a.x),
          y = b.y - float(a.y),
          z = b.z - float(a.z);
    return x * x + y * y + z * z;
}

// близость двух центров классов
__device__ float norm(float4 a, float4 b)
{
    float x = a.x - b.x,
          y = a.y - b.y,
          z = a.z - b.z;
    return x * x + y * y + z * z;
}

__device__ int calc_best_distance(const uchar4& point, const int n_classes)
{
    int best_class = 0;
    float best_distance = 1e15;

    for (int i = 0; i < n_classes; i++) {
        float curr_distance = distance(point, dev_centers[i]);
        // printf("compare %f<>%f\n", curr_distance, best_distance);
        if (curr_distance < best_distance) {
            best_class = i;
            best_distance = curr_distance;
        }
    }

    return best_class;
}

// классификация пикселей по текущим центрам групп.
__global__ void kernel(
    uchar4* data, size_t n,
    float4* new_centers, ulonglong4* cache, uint32_t n_classes,
    unsigned long long* equal)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;

    for (size_t i = id; i < n; i += offset) {
        uchar4& elem = data[i];
        elem.w = calc_best_distance(elem, n_classes);

        // вычисление новых центров классов
        // суммирование значений пикселей по классам
        ulonglong4* cache_elem = &cache[elem.w];
        atomicAdd(&cache_elem->x, elem.x);
        atomicAdd(&cache_elem->y, elem.y);
        atomicAdd(&cache_elem->z, elem.z);
        atomicAdd(&cache_elem->w, 1);
    }

    __syncthreads();
    // присваивание новых значений центров классов.
    for (uint32_t i = id; i < n_classes; i += offset) {
        ulonglong4 cache_elem = cache[i];
        float4 elem = make_float4(
            float(cache_elem.x) / float(cache_elem.w),
            float(cache_elem.y) / float(cache_elem.w),
            float(cache_elem.z) / float(cache_elem.w),
            0.0f);
        new_centers[i] = elem;

        // условие сходимости -- центры не изменились
        float4 old = dev_centers[i];
        if (norm(old, elem) > EPS) {
            // printf("%f %f %f <> %f %f %f\n", old.x, old.y, old.z, elem.x, elem.y, elem.z);
            atomicAdd(equal, 1);
        }
    }
}

__global__ void debug(uchar4* data, const size_t n)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;

    for (size_t i = id; i < n; i += offset) {
        uchar4& p = data[i];
        float4 pp = dev_centers[p.w];
        p.x = (unsigned char)pp.x;
        p.y = (unsigned char)pp.y;
        p.z = (unsigned char)pp.z;
    }
}

typedef struct {
    int x, y;
} Center;

void launch_k_means(uchar4* host_data, const size_t w, const size_t h, const Center* start_centers, const uint32_t n_classes)
{
    const size_t n = h * w;

    uchar4* dev_data;
    // свежевычисленные центры на основе текущего распределения по классам
    float4* dev_next_centers;
    // суммируются все пиксели для вычисления центров классов
    ulonglong4* dev_cache;

    CSC(hipMalloc(&dev_data, sizeof(uchar4) * n));
    CSC(hipMemcpy(dev_data, host_data, sizeof(uchar4) * n, hipMemcpyHostToDevice));

    CSC(hipMalloc(&dev_next_centers, sizeof(float4) * n_classes));
    CSC(hipMalloc(&dev_cache, sizeof(ulonglong4) * n_classes));

    // инициализация центров классов по их координатам
    {
        float4* tmp_centers;
        // значения указанных пикселей.
        tmp_centers = (float4*)malloc(sizeof(float4) * n_classes);
        for (uint32_t i = 0; i < n_classes; i++) {
            uchar4 elem = host_data[(size_t)start_centers[i].y * w + (size_t)start_centers[i].x];
            tmp_centers[i] = make_float4(elem.x, elem.y, elem.z, 0.0f);
        }
        // printf("init\n");
        // for (int i = 0; i < n_classes; i++) {
        //     uchar4 m = tmp_centers[i];
        // printf("%d %d %d\n", m.x, m.y, m.z);
        // }
        // printf("\n\n");
        CSC(hipMemcpy(dev_next_centers, tmp_centers, sizeof(float4) * n_classes, hipMemcpyHostToDevice));
        free(tmp_centers);
    }

    unsigned long long equal = 0,
                       *dev_equal;
    CSC(hipMalloc(&dev_equal, sizeof(unsigned long long)));

    while (true) {
        CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_centers), dev_next_centers, sizeof(float4) * n_classes, 0, hipMemcpyDeviceToDevice));

        // CSC(hipMemcpy(host_data, dev_data, sizeof(uchar4) * n, hipMemcpyDeviceToHost));
        // printf("===\n");
        // for (int x = 0; x < w; x++) {
        //     for (int y = 0; y < h; y++) {
        //         printf("%d ", x * w + y);
        //         uchar4 p = host_data[x * w + y];
        //         printf("%02x%02x%02x %d ", p.x, p.y, p.z, p.w);
        //     }
        //     printf("\n");
        // }
        // printf("===\n");
        // for (int i = 0; i < n; i++) {
        //     printf("%d ", i);
        //     uchar4 p = host_data[i];
        //     printf("%d %d %d %d\n", p.x, p.y, p.z, p.w);
        // }
        // printf("===\n");

        CSC(hipMemset(dev_equal, 0, sizeof(unsigned long long)));
        CSC(hipMemset(dev_cache, 0, sizeof(ulonglong4) * n_classes));

        START_KERNEL((kernel<<<blocks, threads>>>(
            dev_data, n,
            dev_next_centers, dev_cache, n_classes,
            dev_equal)));

        CSC(hipMemcpy(&equal, dev_equal, sizeof(unsigned long long), hipMemcpyDeviceToHost));
        // printf("equal: %llu\n", equal);
        if (equal == 0) {
            break;
        }
    }

    // START_KERNEL((debug<<<blocks, threads>>>(dev_data, n)));

    CSC(hipFree(dev_equal));
    CSC(hipMemcpy(host_data, dev_data, sizeof(uchar4) * n, hipMemcpyDeviceToHost));
    CSC(hipFree(dev_data));
    CSC(hipFree(dev_next_centers));
    CSC(hipFree(dev_cache));
}

int main()
{
#ifdef BENCHMARK
    for (int i = 1; i < argc; i += 2) {
        if (strcmp(argv[i], "-blocks") == 0) {
            blocks = atoi(argv[i + 1]);
        } else if (strcmp(argv[i], "-threads") == 0) {
            threads = atoi(argv[i + 1]);
        }
    }
#endif

    char input[PATH_MAX], output[PATH_MAX];

    scanf("%s", input);
    scanf("%s", output);

    uint32_t n_classes;
    Center* centers;
    scanf("%d", &n_classes);
    centers = (Center*)malloc(n_classes * sizeof(Center));
    for (uint32_t i = 0; i < n_classes; i++) {
        scanf("%d", &centers[i].x);
        scanf("%d", &centers[i].y);
    }

    FILE* in = fopen(input, "rb");
    if (ferror(in)) {
        printf("ERROR opening input file: %s\n", input);
        exit(0);
    }

    uchar4* data;
    uint32_t w, h;
    read_image(in, &data, &w, &h);
    fclose(in);

    launch_k_means(data, h, w, centers, n_classes);

    FILE* out = fopen(output, "wb");
    write_image(out, data, w, h);
    fclose(out);

    free(data);
    free(centers);
    return 0;
}